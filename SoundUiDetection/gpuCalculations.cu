#include "hip/hip_runtime.h"
#include "gpuWrapper.hpp"

__global__ static void crossCorrelateKernel(const float* noise, const float* signal, float* correlation,
    int noiseSize, int signalSize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid <= noiseSize - signalSize) {
        float sum = 0.0f;
        for (int j = 0; j < signalSize; ++j) {
            sum += noise[tid + j] * signal[j];
        }
        correlation[tid] = sum;
    }
}

std::vector<float> correlationGpu(std::vector<float>& signal, std::vector<float>& noise) {
    float* d_signal, * d_noise, * d_correlation;

    hipMalloc((void**)&d_signal, signal.size() * sizeof(float));
    hipMalloc((void**)&d_noise, noise.size() * sizeof(float));
    hipMalloc((void**)&d_correlation, (noise.size() - signal.size() + 1) * sizeof(float));

    hipMemcpy(d_signal, signal.data(), signal.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_noise, noise.data(), noise.size() * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (noise.size() - signal.size() + 1 + blockSize - 1) / blockSize;

    crossCorrelateKernel<<<numBlocks, blockSize >>>(d_noise, d_signal, d_correlation,noise.size(), signal.size());

    std::vector<float> correlation(noise.size() - signal.size() + 1);
    hipMemcpy(correlation.data(), d_correlation, correlation.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_signal);
    hipFree(d_noise);
    hipFree(d_correlation);

    return correlation;
}